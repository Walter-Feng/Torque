#include "hip/hip_runtime.h"
#include "block_sparse.cuh"

#define MAX_RANK 10

namespace torque {
namespace gpu {
namespace block_sparse {
    template<typename T, bool reverse>
    __global__
    void
    reshape_kernel(const T * src_data,
                   const int * block_index_tables,
                   const int * blocks_strides,
                   const int * blocks_offsets,
                   const int * blocks_n_elem_nest_sum,
                   int n_block,
                   int n_elem,
                   int rank,
                   const int * dest_index_table,
                   T * dest_data) {

        const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < n_elem) {
            int block_index = -1;
            int tensor_index = 0;
            int dest_index = 0;
            int tmp;
            int tensor_residue;

            for (int j = 0; j < n_block; j++) {
                if (i >= blocks_n_elem_nest_sum[j]) {
                    block_index += 1;
                } else {
                    block_index += 0;
                }
            }

            tensor_residue = i - blocks_n_elem_nest_sum[block_index];

            for (int j = 0; j < rank; j++) {

                tmp = tensor_residue / block_index_tables[block_index * rank + rank - j - 1];

                tensor_index += blocks_strides[block_index * rank + rank - j - 1] * tmp;
                dest_index += dest_index_table[rank - j - 1] * tmp;

                tensor_residue %= block_index_tables[block_index * rank + rank - j - 1];

            }

            tensor_index += blocks_offsets[block_index];
            dest_index += block_index * dest_index_table[rank];

            if constexpr(reverse) {
                dest_data[tensor_index] = src_data[dest_index];
            } else {
                dest_data[dest_index] = src_data[tensor_index];
            }
        }

    }

    template
    __global__
    void
    reshape_kernel<float, true>(const float * src_data,
                                const int * block_index_tables,
                                const int * blocks_strides,
                                const int * blocks_offsets,
                                const int * blocks_n_elem_nest_sum,
                                int n_block,
                                int n_elem,
                                int rank,
                                const int * dest_index_table,
                                float * dest_data);

    template
    __global__
    void
    reshape_kernel<double, true>(const double * src_data,
                                 const int * block_index_tables,
                                 const int * blocks_strides,
                                 const int * blocks_offsets,
                                 const int * blocks_n_elem_nest_sum,
                                 int n_block,
                                 int n_elem,
                                 int rank,
                                 const int * dest_index_table,
                                 double * dest_data);

    template
    __global__
    void
    reshape_kernel<half, true>(const half * src_data,
                               const int * block_index_tables,
                               const int * blocks_strides,
                               const int * blocks_offsets,
                               const int * blocks_n_elem_nest_sum,
                               int n_block,
                               int n_elem,
                               int rank,
                               const int * dest_index_table,
                               half * dest_data);

    template
    __global__
    void
    reshape_kernel<float, false>(const float * src_data,
                                 const int * block_index_tables,
                                 const int * blocks_strides,
                                 const int * blocks_offsets,
                                 const int * blocks_n_elem_nest_sum,
                                 int n_block,
                                 int n_elem,
                                 int rank,
                                 const int * dest_index_table,
                                 float * dest_data);

    template
    __global__
    void
    reshape_kernel<double, false>(const double * src_data,
                                  const int * block_index_tables,
                                  const int * blocks_strides,
                                  const int * blocks_offsets,
                                  const int * blocks_n_elem_nest_sum,
                                  int n_block,
                                  int n_elem,
                                  int rank,
                                  const int * dest_index_table,
                                  double * dest_data);

    template
    __global__
    void
    reshape_kernel<half, false>(const half *src_data,
                                const int * block_index_tables,
                                const int * blocks_strides,
                                const int * blocks_offsets,
                                const int * blocks_n_elem_nest_sum,
                                int n_block,
                                int n_elem,
                                int rank,
                                const int * dest_index_table,
                                half * dest_data);
}
}
}
